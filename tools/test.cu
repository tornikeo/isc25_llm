#include <cstdio>
#include <hip/hip_runtime.h>

__global__ void testKernel() {
    // Do nothing
}

int main() {
    // Launch kernel with 1 block and 1 thread
    testKernel<<<1, 1>>>();

    // Check for launch errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Kernel launch failed: %s\n", hipGetErrorString(err));
        return 1;
    }

    // Wait for kernel to finish and check for errors
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        printf("Kernel execution failed: %s\n", hipGetErrorString(err));
        return 1;
    }

    printf("CUDA kernel executed successfully.\n");
    return 0;
}